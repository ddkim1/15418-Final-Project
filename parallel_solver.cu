#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <getopt.h>
#include <string>
#include <cstring>
#include "minesweeper.h" 

#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#include <driver_functions.h> 

#include "CycleTimer.h" 

#define PART_SIDE 3 

__device__ bool 
cu_is_valid(int y, int x, int height, int width) { 
    return (y >= 0 && y < height && x >= 0 && x < width); 
} 

__device__ int 
cu_neighboring_mines(int y, int x, int* gpuSolverBoard, int height, int width) { 
    int neighboringmines = 0; 
    for (int dy = -1; dy < 2; dy++) { 
        for (int dx = -1; dx < 2; dx++) { 
            if (cu_is_valid(y + dy, x + dx, height, width) && gpuSolverBoard[(y + dy) * width + (x + dx)] == -1) { 
                // printf("blockIdx.x: %d\n", blockIdx.x); 
                neighboringmines++; 
            }
        }
    } 
    return neighboringmines; 
} 

__device__ int 
cu_unknown_tiles(int y, int x, int* gpuSolverBoard, int height, int width) { 
    int unknownTiles = 0; 
    for (int dy = -1; dy < 2; dy++) { 
        for (int dx = -1; dx < 2; dx++) { 
            if (cu_is_valid(y + dy, x + dx, height, width) && gpuSolverBoard[(y + dy) * width + (x + dx)] == -10) { 
                unknownTiles++; 
            } 
        } 
    } 
    return unknownTiles; 
} 

__device__ void 
cu_double_tap(int y, int x, int* gpuSolverBoard, int* tempSolverBoard, int height, int width, int* gpuBoard) { 
    for (int dy = -1; dy < 2; dy++) { 
        for (int dx = -1; dx < 2; dx++) { 
            if (cu_is_valid(y + dy, x + dx, height, width)) { 
                // printf("revealing blockIdx %d global_square_idx_y %d global_square_idx_x %d\n", blockIdx.x, y + dy, x + dx); 
                tempSolverBoard[(height * width * blockIdx.x) + (y + dy) * width + (x + dx)] = gpuBoard[(y + dy) * width + (x + dx)]; 
            } 
        }
    }
} 

__device__ void 
cu_deduce_mines(int y, int x, int* tempSolverBoard, int height, int width, int* gpuSolverMineLocations, int* cudaCountMineFound) { 
    for (int dy = -1; dy < 2; dy++) { 
        for (int dx = -1; dx < 2; dx++) { 
            if (cu_is_valid(y + dy, x + dx, height, width) && tempSolverBoard[(height * width * blockIdx.x) + (y + dy) * width + (x + dx)] == -10) { 
                // printf("revealing bomb at blockIdx %d global_square_idx_y %d global_square_idx_x %d\n", blockIdx.x, y + dy, x + dx); 
                tempSolverBoard[(height * width * blockIdx.x) + (y + dy) * width + (x + dx)] = -1; 
                int updated_idx = atomicAdd(cudaCountMineFound, 1); 
                gpuSolverMineLocations[updated_idx * 2] = y + dy; 
                gpuSolverMineLocations[updated_idx * 2 + 1] = x + dx; 
            } 
        } 
    } 
}

__global__ void 
moveDoubleTap_kernel(int* gpuBoard, 
                     int* gpuSolverBoard, 
                     int* gpuMineLocations, 
                     int* gpuSolverMineLocations, 
                     int height, 
                     int width, 
                     int part_length_x, 
                     int part_length_y, 
                     int* cudaCountMineFound, 
                     int* tempSolverBoard) { 
    // this actually perform both randomMove and the double tap function 
    // here, we first find the next randomly guessing entry 
    __shared__ int this_round_guess_pos[2]; 

    // int local_num_failure = 0; 
    int local_part_h_idx = blockIdx.x/part_length_x; 
    int local_part_v_idx = blockIdx.x%part_length_x; 
    int local_square_offset_y = local_part_h_idx * PART_SIDE; 
    int local_square_offset_x = local_part_v_idx * PART_SIDE; 
    bool backtrace = false; 
    /*
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x; 
    if (global_idx == 0) { 
        printf("got in\n"); 
        for (int i = 0; i < 10; i++) { 
            printf("%d\n", gpuSolverBoard[i]); 
        }
    }
    __syncthreads(); 
    */ 

    // printf("blockIdx.x %d, threadIdx.x %d, local_part_h_idx %d, local_part_v_idx %d, local_square_offset_y %d, local_square_offset_x %d\n", blockIdx.x, threadIdx.x, local_part_h_idx, local_part_v_idx, local_square_offset_y, local_square_offset_x); 
    // printf("lots of print\n"); 
    if (threadIdx.x == 0) { 
        int randY = 0; 
        int randX = 0; 
        // one thread of each process would be used to determine which square in the corresponding part will be appointed. 
        for (int i = 0; i < PART_SIDE; i++)  { // y 
            for (int j = 0; j < PART_SIDE; j++) { // x 
                int global_square_idx_y = i + local_square_offset_y; 
                int global_square_idx_x = j + local_square_offset_x; 
                // if (blockIdx.x == 0 || blockIdx.x == 1) 
                    // printf("we can discuss implementation of block idx %d global_sq_y %d, global_sq_x\n", blockIdx.x, global_square_idx_x, global_square_idx_x); 
                if (gpuSolverBoard[global_square_idx_y * width + global_square_idx_x] == -10) { 
                    randY = global_square_idx_y;  // we use the local grid idx 
                    randX = global_square_idx_x;  // local grid idx 
                    // printf("blockIdx.x %d threadIdx.x %d, randY is %d, randX is %d\n", blockIdx.x, threadIdx.x, randY, randX); 
                    // break; 
                }
            }
        } 
        this_round_guess_pos[0] = randY; 
        this_round_guess_pos[1] = randX; 
        // then, we run through the performMove functionality 
        gpuSolverBoard[randY * width + randX] = gpuBoard[randY * width + randX]; 
        tempSolverBoard[(height * width * blockIdx.x) + randY * width + randX] = gpuBoard[randY * width + randX]; 

        if (gpuBoard[randY * width + randX] == -1) { 
            int updated_idx = atomicAdd(cudaCountMineFound, 1); 
            gpuSolverMineLocations[updated_idx * 2] = randY; 
            gpuSolverMineLocations[updated_idx * 2 + 1] = randX; 
            // printf("blockIdx.x %d random click is a bomb\n", blockIdx.x); 
        } 
    }
    __syncthreads(); 
    // printf("blockIdx.x %d, we have the chosen idx to be %d %d\n", blockIdx.x, this_round_guess_pos[0], this_round_guess_pos[1]); 

    // identify which square does each thread take charge of 
    int local_idx_y = threadIdx.x / PART_SIDE; 
    int local_idx_x = threadIdx.x % PART_SIDE; 
    int global_square_idx_y = local_idx_y + local_square_offset_y; 
    int global_square_idx_x = local_idx_x + local_square_offset_x; 
    int randY = this_round_guess_pos[0]; 
    int randX = this_round_guess_pos[1]; 
    // printf("blockIdx.x %d, threadIdx.x %d, global_square_idx_y %d, global_square_idx_x %d, randY %d, randX %d\n", blockIdx.x, threadIdx.x, global_square_idx_y, global_square_idx_x, randY, randX); 

    // here, we are left with block where they found that a new starting point has emerged 
    int neighborMines = cu_neighboring_mines(randY, randX, gpuSolverBoard, height, width); 
    int unknownSquares = cu_unknown_tiles(randY, randX, gpuSolverBoard, height, width); 
    // printf("blockIdx.x %d, threadIdx.x %d, neighborMines %d, unknownSquares %d\n", blockIdx.x, threadIdx.x, neighborMines, unknownSquares); 
    // if (blockIdx.x == 0 && threadIdx.x == 0) { 
        // printf("critical location bomb: %d\n", gpuSolverBoard[randY * width + randX]); 
    // }

    // double tap 
    if (neighborMines == gpuSolverBoard[randY * width + randX] || gpuSolverBoard[randY * width + randX] == 0) { 
        // double tap but using multi-thread 
        // printf("double tap situation is found for blockidx.x %d and randY %d randX %d\n", blockIdx.x, randY, randX); 
        // if (global_square_idx_y >= randY - 1 && global_square_idx_y < randY + 2 && global_square_idx_x >= randX - 1 && global_square_idx_x < randX + 2) { 
        //     printf("revealing blockIdx %d global_square_idx_y %d global_square_idx_x %d\n", blockIdx.x, global_square_idx_y, global_square_idx_x); 
        //     tempSolverBoard[(height * width * blockIdx.x) + (global_square_idx_y * width + global_square_idx_x)] = gpuBoard[global_square_idx_y * width + global_square_idx_x]; 
        // } 
        if (threadIdx.x == 0) { 
            cu_double_tap(randY, randX, gpuSolverBoard, tempSolverBoard, height, width, gpuBoard); 
        } 
        backtrace = true; 
    } 
    
    __syncthreads(); 

    // deduce mines 
    if (unknownSquares == gpuSolverBoard[randY * width + randX] - neighborMines) { 
        // printf("deduce mines situation is found for blockidx.x %d and randY %d randX %d\n", blockIdx.x, randY, randX); 
        // if (global_square_idx_y >= randY - 1 && global_square_idx_y < randY + 2 && global_square_idx_x >= randX - 1 && global_square_idx_x < randX + 2) { 
        //     if (gpuSolverBoard[global_square_idx_y * width + global_square_idx_x] == -10) { 
        //         printf("revealing bomb at blockIdx %d global_square_idx_y %d global_square_idx_x %d\n", blockIdx.x, global_square_idx_y, global_square_idx_x); 
        //         tempSolverBoard[(height * width * blockIdx.x) + (global_square_idx_y * width + global_square_idx_x)] = -1; 
        //         int updated_idx = atomicAdd(cudaCountMineFound, 1); 
        //         gpuSolverMineLocations[updated_idx * 2] = global_square_idx_y; 
        //         gpuSolverMineLocations[updated_idx * 2 + 1] = global_square_idx_x; 
        //         printf("all of them are bombs\n"); 
        //     } 
        // } 
        if (threadIdx.x == 0) { 
            cu_deduce_mines(randY, randX, tempSolverBoard, height, width, gpuSolverMineLocations, cudaCountMineFound); 
        } 
        backtrace = true; 
    } 
    __syncthreads(); 

    // printf("got after the heat check\n"); 

    return; 
} 
    
    
    // if (tempSolverBoard[(height * width * blockIdx.x) + global_square_idx_y * width + global_square_idx_x] != -10) { 
    //     // not occupied in the current block, but may be modified in other blocks 
    //     for (int i = 0; i < gridDim.x; i++) { 
    //         if (i == blockIdx.x) { 
    //             continue; 
    //         }
    //         if (tempSolverBoard[(height * width * i) + global_square_idx_y * width + global_square_idx_x] == -10) { 
    //             tempSolverBoard[(height * width * i) + global_square_idx_y * width + global_square_idx_x] = tempSolverBoard[(height * width * blockIdx.x) + global_square_idx_y * width + global_square_idx_x]; 
    //         } 
    //     } 
    // } 

    // if (blockIdx.x == 3 && threadIdx.x == 2) { 
    //     for (int i = 0; i < gridDim.x; i++) { 
    //         printf("global_square_idx_y %d, i %d, global_square_idx_x %d, %d\n", global_square_idx_y, i, global_square_idx_x, tempSolverBoard[(height * width * i) + global_square_idx_y * global_square_idx_x]); 
    //     }
    // } 

    // return; 

__global__ void 
backtracing_kernel(int* gpuBoard, 
                     int* gpuSolverBoard, 
                     int* gpuMineLocations, 
                     int* gpuSolverMineLocations, 
                     int height, 
                     int width, 
                     int part_length_x, 
                     int part_length_y, 
                     int* cudaCountMineFound, 
                     int* tempSolverBoard, 
                     int* outcome) { 
    
    // bool backtrace = true; 
    int backtrace = 0; 
    int neighborMines; 
    int unknownSquares; 

    int local_part_h_idx = blockIdx.x/part_length_x; 
    int local_part_v_idx = blockIdx.x%part_length_x; 
    int local_square_offset_y = local_part_h_idx * PART_SIDE; 
    int local_square_offset_x = local_part_v_idx * PART_SIDE; 
    int local_idx_y = threadIdx.x / PART_SIDE; 
    int local_idx_x = threadIdx.x % PART_SIDE; 
    int global_square_idx_y = local_idx_y + local_square_offset_y; 
    int global_square_idx_x = local_idx_x + local_square_offset_x; 
    // int randY = this_round_guess_pos[0]; 
    // int randX = this_round_guess_pos[1]; 

    // big while loop is here 
    // backtrace = false; 
    if (tempSolverBoard[(height * width * blockIdx.x) + global_square_idx_y * width + global_square_idx_x] == -10) { 
        return; 
    } 
    neighborMines = cu_neighboring_mines(global_square_idx_y, global_square_idx_x, gpuSolverBoard, height, width); 
    unknownSquares = cu_unknown_tiles(global_square_idx_y, global_square_idx_x, gpuSolverBoard, height, width); 
    if (unknownSquares == 0) { 
        return; 
    } 
    if (neighborMines == tempSolverBoard[(height * width * blockIdx.x) + global_square_idx_y * width + global_square_idx_x] || tempSolverBoard[(height * width * blockIdx.x) + global_square_idx_y * width + global_square_idx_x] == 0) { 
        cu_double_tap(global_square_idx_y, global_square_idx_x, gpuSolverBoard, tempSolverBoard, height, width, gpuBoard); 
        backtrace = 1; 
    } 
    if (unknownSquares == tempSolverBoard[(height * width * blockIdx.x) + global_square_idx_y * width + global_square_idx_x] - neighborMines) { 
        cu_deduce_mines(global_square_idx_y, global_square_idx_x, tempSolverBoard, height, width, gpuSolverMineLocations, cudaCountMineFound); 
        backtrace = 1; 
    } 
        // if (tempSolverBoard[(height * width * blockIdx.x) + global_square_idx_y * width + global_square_idx_x] == -10) { 
        //     // not occupied in the current block, but may be modified in other blocks 
        //     for (int i = 0; i < blockDim.x; i++) { 
        //         if (i == blockIdx.x) { 
        //             continue; 
        //         }
        //         if (tempSolverBoard[(height * width * i) + global_square_idx_y * width + global_square_idx_x] != -10) { 
        //             tempSolverBoard[(height * width * blockIdx.x) + global_square_idx_y * width + global_square_idx_x] = tempSolverBoard[(height * width * i) + global_square_idx_y * width + global_square_idx_x]; 
        //         } 
        //     } 
        // } 
    *outcome = backtrace; 
} 

__global__ void 
update_gpu_solver_board(int* gpuSolverBoard, int* tempSolverBoard, int height, int width, int num_of_block) { 
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    if (idx >= height * width) { 
        return; 
    } 
    int tempSolverEventualValue = -10; 
    for (int i = 0; i < num_of_block; i++) { 
        int idx_temp = idx + (i * height * width); 
        int tempSolverEntry = tempSolverBoard[idx_temp]; 
        if (gpuSolverBoard[idx] != tempSolverEntry && tempSolverEntry != -10) { 
            gpuSolverBoard[idx] = tempSolverEntry; 
            tempSolverEventualValue = tempSolverEntry; 
        } 
    } 
    if (tempSolverEventualValue != -10) { 
        for (int i = 0; i < num_of_block; i++) { 
            int idx_temp = idx + (i * height * width); 
            tempSolverBoard[idx_temp] = tempSolverEventualValue; 
        }
    } 
    return; 
}

void Minesweeper::parallel_cuda_solver() { 
    if (height % PART_SIDE != 0 || width % PART_SIDE != 0) { 
        // printf("got inside the exit statement if\n"); 
        exit(0); 
    } 
    int num_of_blocks = (height/PART_SIDE) * (width/PART_SIDE); 
    const int threadsPerBlock = PART_SIDE * PART_SIDE; 
    int part_length_y = height/PART_SIDE; 
    int part_length_x = width/PART_SIDE; 
    // printf("number_of_blocks %d, threadsPerBlock %d, part_length_y %d, part_length_x %d\n", num_of_blocks, threadsPerBlock, part_length_x, part_length_y); 

    // GPU might be hard to work with when the array allocated is 2D 
    int* gpuBoard; 
    int* gpuSolverBoard; 
    int* gpuMineLocations; 
    int* gpuSolverMineLocations; 
    // printf("got here inside the cuda parallel\n"); 

    // allocating memory spaces on the GPU 
    hipMalloc(&gpuBoard, sizeof(int) * height * width); 
    hipMalloc(&gpuSolverBoard, sizeof(int) * height * width); 
    hipMalloc(&gpuMineLocations, sizeof(int) * 2 * mines); 
    hipMalloc(&gpuSolverMineLocations, sizeof(int) * 2 * mines); 

    // getting information onto the GPU actually 

    board2 = (int*)malloc(height * width * sizeof(int)); 
    solverboard2 = (int*)malloc(height * width * sizeof(int)); 
    for (int i = 0; i < height; i++) { 
        for (int j = 0; j < width; j++) { 
            board2[i * width + j] = board[i][j]; 
            solverboard2[i * width + j] = solverboard[i][j]; 
        }
    }
    
    hipMemcpy(gpuBoard, board2, sizeof(int) * height * width, hipMemcpyHostToDevice); 
    hipMemcpy(gpuSolverBoard, solverboard2, sizeof(int) * height * width, hipMemcpyHostToDevice); 
    hipMemcpy(gpuMineLocations, mineLocations, sizeof(int) * 2 * mines, hipMemcpyHostToDevice); 
    hipMemcpy(gpuSolverMineLocations, solverMineLocations, sizeof(int) * 2 * mines, hipMemcpyHostToDevice); 
    
    // strange num to keep track of minesLeft 
    int* cudaCountMineFound; 
    hipMalloc(&cudaCountMineFound, sizeof(int)); 
    hipMemset(cudaCountMineFound, -1, sizeof(int)); 
    // for (int i = 0; i < 10; i++) { 
    //     printf("%d  ", cudaCountMineFound[i]); 
    // }
    // printf("\n"); 

    // to facilitate calling the kernel, we have to have a holder of all the changes 
    int *tempSolverBoard; 
    hipMalloc(&tempSolverBoard, sizeof(int) * num_of_blocks * height * width); 
    // use a GPT4 generated kernel to copy from gpuSolverBoard to tempSolverBoard 
    for (int i = 0; i < num_of_blocks; i++) { 
        hipMemcpy(tempSolverBoard + (i * height * width), gpuSolverBoard, (height * width * sizeof(int)), hipMemcpyDeviceToDevice); 
    } 

    // printf("got here to finish some memory allocation\n"); 

    // start goinginto the main while loop 

    int* backtrace; 
    int hostbacktrace = 1; 
    hipMalloc(&backtrace, sizeof(int)); 
    hipMemset(backtrace, 0, sizeof(int)); 
    int minesFound = 0; 
    while (minesFound < mines) {
        // launching kernel inside 
        // second kernel launches num_of_blocks blocks and each part_side squared threads, do double tap 
        moveDoubleTap_kernel<<<num_of_blocks, threadsPerBlock>>>(gpuBoard, gpuSolverBoard, gpuMineLocations, gpuSolverMineLocations, height, width, part_length_x, part_length_y, cudaCountMineFound, tempSolverBoard); 
        hipDeviceSynchronize(); 

        hipMemset(backtrace, 0, sizeof(int)); 
        int temp_block = (height * width + 256 - 1)/256; 
        while (hostbacktrace == 1) { 
            update_gpu_solver_board<<<temp_block, 256>>>(gpuSolverBoard = gpuSolverBoard, tempSolverBoard = tempSolverBoard, height = height, width = width, num_of_blocks = num_of_blocks); 
            hipDeviceSynchronize(); 
            backtracing_kernel<<<num_of_blocks, threadsPerBlock>>>(gpuBoard, gpuSolverBoard, gpuMineLocations, gpuSolverMineLocations, height, width, part_length_x, part_length_y, cudaCountMineFound, tempSolverBoard, backtrace); 
            hipMemcpy(&hostbacktrace, backtrace, sizeof(int), hipMemcpyDeviceToHost); 
        } 

        hipMemcpy(&minesFound, cudaCountMineFound, sizeof(int), hipMemcpyDeviceToHost); 
        // printf("********** %d\n", minesFound); 
        // printf("total mines: %d\n", mines); 

        // prepare for the next round 
        update_gpu_solver_board<<<temp_block, 256>>>(gpuSolverBoard = gpuSolverBoard, tempSolverBoard = tempSolverBoard, height = height, width = width, num_of_blocks = num_of_blocks); 
    } 
    minesLeft = mines - minesFound; 
    // printf("got here!!!!!\n"); 
    // hipMemcpy(mineLocations, gpuMineLocations, sizeof(int) * 2 * mines, hipMemcpyDeviceToHost); 
    hipMemcpy(solverMineLocations, gpuSolverMineLocations, sizeof(int) * 2 * mines, hipMemcpyDeviceToHost); 
    hipMemcpy(solverboard2, gpuSolverBoard, sizeof(int) * 2 * mines, hipMemcpyDeviceToHost); 
    // printf("got here!!!!!\n"); 

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    } 
    // printf("got here!!!!!\n"); 

    hipFree(gpuBoard); 
    hipFree(gpuSolverBoard); 
    // printf("got here!!!!!\n"); 
    hipFree(gpuMineLocations); 
    hipFree(gpuSolverMineLocations); 
    // printf("got here!!!!!\n"); 
}
